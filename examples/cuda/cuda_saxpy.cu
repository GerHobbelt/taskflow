#include "hip/hip_runtime.h"
// This program performs a simple single-precision Ax+Y operation
// using cudaFlow and verifies its result.

#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

// Kernel: saxpy
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

// Function: main
int main() {
  
  const unsigned N = 1<<20;

  std::vector<float> hx, hy;

  float* dx {nullptr};
  float* dy {nullptr};
  
  // allocate x
  hx.resize(N, 1.0f);
  hipMalloc(&dx, N*sizeof(float));

  // allocate y
  hy.resize(N, 2.0f);
  hipMalloc(&dy, N*sizeof(float));
  
  // saxpy cudaFlow
  tf::cudaFlow cf;
  auto h2d_x = cf.copy(dx, hx.data(), N);
  auto h2d_y = cf.copy(dy, hy.data(), N);
  auto d2h_x = cf.copy(hx.data(), dx, N);
  auto d2h_y = cf.copy(hy.data(), dy, N);
  auto kernel = cf.kernel((N+255)/256, 256, 0, saxpy, N, 2.0f, dx, dy);
  kernel.succeed(h2d_x, h2d_y)
        .precede(d2h_x, d2h_y);
  
  tf::cudaStream stream;
  tf::cudaGraphExec exec(cf);
  exec.run(stream);
  stream.synchronize();
  
  // visualize this cudaflow
  cf.dump(std::cout);

  // Add a verification task
  float max_error = 0.0f;
  for (size_t i = 0; i < N; i++) {
    max_error = std::max(max_error, abs(hx[i]-1.0f));
    max_error = std::max(max_error, abs(hy[i]-4.0f));
  }
  std::cout << "saxpy finished with max error: " << max_error << '\n';

  // free memory
  hipFree(dx);
  hipFree(dy);

  return 0;
}

